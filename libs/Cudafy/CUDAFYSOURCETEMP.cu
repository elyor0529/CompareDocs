
#include <hip/hip_runtime.h>

// CudafyModuleViewer.CUDACheck
extern "C" __global__  void TestKernelFunction( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);

// CudafyModuleViewer.CUDACheck
extern "C" __global__  void TestKernelFunction( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	int num = threadIdx.x + blockIdx.x * blockDim.x;
	c[(num)] = a[(num)] + b[(num)];
}
